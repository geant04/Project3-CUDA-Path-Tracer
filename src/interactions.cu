#include "hip/hip_runtime.h"
#include "interactions.h"

#include "utilities.h"

#include <thrust/random.h>

__host__ __device__ glm::vec3 calculateRandomDirectionInHemisphere(
    glm::vec3 normal,
    thrust::default_random_engine &rng)
{
    thrust::uniform_real_distribution<float> u01(0, 1);

    float up = sqrt(u01(rng)); // cos(theta)
    float over = sqrt(1 - up * up); // sin(theta)
    float around = u01(rng) * TWO_PI;

    // Find a direction that is not the normal based off of whether or not the
    // normal's components are all equal to sqrt(1/3) or whether or not at
    // least one component is less than sqrt(1/3). Learned this trick from
    // Peter Kutz.

    glm::vec3 directionNotNormal;
    if (abs(normal.x) < SQRT_OF_ONE_THIRD)
    {
        directionNotNormal = glm::vec3(1, 0, 0);
    }
    else if (abs(normal.y) < SQRT_OF_ONE_THIRD)
    {
        directionNotNormal = glm::vec3(0, 1, 0);
    }
    else
    {
        directionNotNormal = glm::vec3(0, 0, 1);
    }

    // Use not-normal direction to generate two perpendicular directions
    glm::vec3 perpendicularDirection1 =
        glm::normalize(glm::cross(normal, directionNotNormal));
    glm::vec3 perpendicularDirection2 =
        glm::normalize(glm::cross(normal, perpendicularDirection1));

    return up * normal
        + cos(around) * over * perpendicularDirection1
        + sin(around) * over * perpendicularDirection2;
}

__host__ __device__ glm::vec3 calculateWalterGGXSampling(
    glm::vec3 normal,
    float roughness,
    thrust::default_random_engine &rng)
{
    //// https://www.graphics.cornell.edu/~bjw/microfacetbsdf.pdf
    //// Eq. 35, 36 for GGX sampling distribution
    thrust::uniform_real_distribution<float> u01(0.0f, 1.0f);
    float u0 = u01(rng);
    float u1 = u01(rng);

    float alpha = roughness * roughness;
    float alpha2 = alpha * alpha;

    float cosTheta = sqrt((1.0f - u0) / (u0 * (alpha2 - 1.0f) + 1.0f)); 
    float sinTheta = sqrt(1 - cosTheta * cosTheta);
    float phi = TWO_PI * u1;

    glm::vec3 directionNotNormal;
    if (abs(normal.x) < SQRT_OF_ONE_THIRD)
    {
        directionNotNormal = glm::vec3(1, 0, 0);
    }
    else if (abs(normal.y) < SQRT_OF_ONE_THIRD)
    {
        directionNotNormal = glm::vec3(0, 1, 0);
    }
    else
    {
        directionNotNormal = glm::vec3(0, 0, 1);
    }

    glm::vec3 perpendicularDirection1 =
        glm::normalize(glm::cross(normal, directionNotNormal));
    glm::vec3 perpendicularDirection2 =
        glm::normalize(glm::cross(normal, perpendicularDirection1));

    return cosTheta * normal
        + cos(phi) * sinTheta * perpendicularDirection1
        + sin(phi) * sinTheta * perpendicularDirection2;
}



// MASSIVE RAY SAMPLING FUNCTION!!!!!!!!!!!!!!!
__host__ __device__ void sampleRay(
    PathSegment & pathSegment,
    glm::vec3 intersect,
    glm::vec3 normal,
    const Material &m,
    thrust::default_random_engine &rng)
{
    glm::vec3 inDirection = glm::normalize(pathSegment.ray.direction);
    glm::vec3 outDirection;
    glm::vec3 halfVector = glm::normalize(inDirection + normal);

    thrust::uniform_real_distribution<float> u01(0.0f, 1.0f);

    // Using Joe Schutte's Disney implementation for this
    float metallicWeight = m.metallic;
    float diffuseWeight = (1.0f - metallicWeight);
    float specularWeight = metallicWeight + diffuseWeight;

    float invSumWeight = 1.0f;

    float pDiffuse = diffuseWeight * invSumWeight;
    // float pSpecular = specularWeight * invSumWeight;

    glm::vec3 F0 = glm::mix(glm::vec3(0.04f), m.color, m.metallic);
    float pSpecular = glm::clamp((F0.x + F0.y + F0.z) / 3.0f, 0.02f, 0.98f);

    float lobeProbability;

    float p = u01(rng);
    glm::vec3 brdf = m.color;

    glm::vec3 wo = -inDirection;
    glm::vec3 wi;
    glm::vec3 diffuse_wi = glm::normalize(calculateRandomDirectionInHemisphere(normal, rng));
    glm::vec3 diffuseNormal = dot(normal, diffuse_wi) < 0.0f ? -diffuse_wi : diffuse_wi;

    if (m.hasReflective)
    {
        // Specular GGX
        glm::vec3 microNormal = glm::normalize(calculateWalterGGXSampling(normal, m.roughness, rng));
        pathSegment.microNormal = microNormal;

        glm::vec3 specularDir = glm::reflect(inDirection, microNormal);

        glm::vec3 F0 = glm::mix(glm::vec3(0.04f), m.color, m.metallic);
        float avgF0 = glm::clamp((F0.x + F0.y + F0.z) / 3.0f, 0.02f, 0.98f);
        bool isSpecularBounce = p < avgF0;

        wi = glm::mix(
            diffuse_wi, 
            specularDir, 
            isSpecularBounce);
        brdf = glm::mix(
            diffuseBRDF(wo, wi, normal, m) / (1.0f - avgF0), 
            specularBRDF(wo, wi, normal, microNormal, m) / avgF0, 
            isSpecularBounce);
    }
    else if (m.hasRefractive)
    {
        glm::vec3 diffuseWi = glm::mix(normal, diffuseNormal, m.roughness);

        float cosThetaI = dot(normal, wo);        
        float etaA = 1.0f;
        float etaB = 1.55f;


        float rand = u01(rng);
        if (rand < 0.50f)
        {
            wi = glm::reflect(glm::normalize(-wo), normal);
            wi = glm::mix(wi, diffuseWi, m.roughness);
            
            // awesome artificial roughness trick from seb. lague
            
            float cosTheta = abs(dot(normal, wi));
            glm::vec3 R0 = glm::vec3((etaA - etaB) / (etaA + etaB));
            R0 = R0 * R0;
            glm::vec3 F = fresnelSchlick(R0, abs(cosThetaI));

            brdf = 2.0f * F;
        }
        else
        {
            // Transmissive material, use the specularBTDF
            float cosThetaI = dot(normal, wo);
            bool entering = cosThetaI > 0.0f;

            float eta = etaA / etaB;
            float iorRatio;

            if (entering)
            {
                iorRatio = eta;
            }
            else
            {
                iorRatio = 1.0f / eta;
            }

            wi = glm::refract(inDirection, (entering) ? normal : -normal, iorRatio);
            wi = glm::mix(wi, (entering) ? -diffuseNormal : diffuseNormal, m.roughness);

            // float F = fresnelDielectric(abs(cosThetaI), etaA, etaB);
            float cosTheta = abs(dot(normal, wi));
            glm::vec3 R0 = glm::vec3((etaA - etaB) / (etaA + etaB));
            R0 = R0 * R0;
            glm::vec3 F = fresnelSchlick(R0, abs(cosThetaI));

            brdf = 2.0f * glm::vec3(1.0f - F);
        }
    }
    else
    {
        // Sample diffuse
        wi = diffuse_wi;
        brdf = m.color * diffuseBRDF(wo, wi, normal, m);
    }

    // Assign wi
    pathSegment.ray.direction = wi;

    // Assign intersect for the next bounce
    pathSegment.ray.origin = intersect + wi * 0.005f;

    // Yeah
    pathSegment.color *= brdf;

    pathSegment.remainingBounces -= 1;
}

__host__ __device__ float GGXDistribution(float alpha, float cosTheta)
{
    // https://agraphicsguynotes.com/posts/sample_microfacet_brdf/#one-extra-step
    float alpha2 = alpha * alpha;
    float cos2Theta = cosTheta * cosTheta;
    float denom = (alpha2 - 1.0f) * cos2Theta + 1.0f;

    return alpha2 / (PI * denom * denom);
}

__host__ __device__ float SmithGGX(
    float nDotI,
    float nDotO,
    float a2
)
{
    // Based on implementation from https://schuttejoe.github.io/post/ggximportancesamplingpart1/,
    // https://media.gdcvault.com/gdc2017/Presentations/Hammon_Earl_PBR_Diffuse_Lighting.pdf, this too maybe.
    // This combines SmithGGX(i, m) * SmithGGX(o, m)
    float denomIn = nDotI * sqrt(a2 + (1.0f - a2) * (nDotI * nDotI));
    float denomOut = nDotO * sqrt(a2 + (1.0f - a2) * (nDotO * nDotO));

    return 2.0f * nDotI * nDotO / (denomIn + denomOut);
}

__host__ __device__ glm::vec3 fresnelSchlick(glm::vec3 F0, float cosTheta)
{
    return F0 + (glm::vec3(1.0f) - F0) * powf(1.0f - cosTheta, 5.0f);
}

// Ripped straight from PBRT 3ed, thanks Google
__host__ __device__ float fresnelDielectric(float cosThetaI, float etaI, float etaT)
{
    cosThetaI = glm::clamp(cosThetaI, -1.0f, 1.0f);

    // Potentially swap indices of refraction
    if(cosThetaI < 0.0f) {
        std::swap(etaI, etaT);
        cosThetaI = -cosThetaI;
    }

    // Compute cosTheta using Snell's law
    float sinThetaI = glm::sqrt(glm::max(0.0f, 1.0f - cosThetaI * cosThetaI));
    float sinThetaT = etaI / etaT * sinThetaI;

    // Check for total internal reflection
    if(sinThetaT >= 1) {
        return 1;
    }

    float cosThetaT = glm::sqrt(glm::max(0.0f, 1.0f - sinThetaT * sinThetaT));

    float Rparl = ((etaT * cosThetaI) - (etaI * cosThetaT)) / ((etaT * cosThetaI) + (etaI * cosThetaT));
    float Rperp = ((etaI * cosThetaI) - (etaT * cosThetaT)) / ((etaI * cosThetaI) + (etaT * cosThetaT));
    return (Rparl * Rparl + Rperp * Rperp) / 2;
}

__host__ __device__ glm::vec3 specularBTDF(
    glm::vec3 wo,
    glm::vec3 wi,
    glm::vec3 normal,
    const Material &m
)
{
    float cosThetaWo = dot(normal, wo);
    bool entering = cosThetaWo > 0;

    return glm::vec3(0.0f, 1.0f, 0.0f);
}


__host__ __device__ glm::vec3 specularBRDF(
    glm::vec3 wo,
    glm::vec3 wi,
    glm::vec3 normal,
    glm::vec3 microNormal,
    const Material &m
)
{
    glm::vec3 half = glm::normalize(wo + wi);
        
    float nDotH = glm::max(dot(normal, half), 0.0f);
    float nDotI = glm::max(dot(normal, wo), 0.0f);
    float nDotO = glm::max(dot(normal, wi), 0.0f);
    float mDotI = glm::max(dot(microNormal,wi), 0.0f);

    if (dot(wi, microNormal) < 0.0f)
    {
        mDotI = -mDotI;
    }

    float G = SmithGGX(nDotI, nDotO, m.roughness);
       
    glm::vec3 albedo = m.color;
    glm::vec3 F0 = glm::mix(glm::vec3(0.04f), albedo, m.metallic);
    
    glm::vec3 metallicF = fresnelSchlick(F0, glm::max(dot(wi, half), 0.0f));
    // I hard coded the IOR of plastic,which is 1.460, into the third arg of fresnelDielectric
    glm::vec3 dielectricF = glm::clamp(glm::vec3(fresnelDielectric(mDotI, 1.0f, 1.460f)), 0.0f, 1.0f); 

    glm::vec3 F = glm::mix(dielectricF, metallicF, m.metallic);

    // Adapted this from Schutte's specular BRDF simplification
    // I'm assuming the D term isn't here because of cut terms from fully evaluating GGX,
    // but I'm confused about why the denominator just doesn't exist in this implementation.
    return F * G;
}

__host__ __device__ glm::vec3 diffuseBRDF(
    glm::vec3 wo,
    glm::vec3 wi,
    glm::vec3 normal,
    const Material &m
)
{
    return m.color; //glm::clamp(m.color, glm::vec3(0.0f), glm::vec3(1.0f));
}